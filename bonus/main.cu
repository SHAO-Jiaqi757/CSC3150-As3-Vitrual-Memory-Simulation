#include "hip/hip_runtime.h"
﻿#include "virtual_memory.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <inttypes.h>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <string.h>

#define DATAFILE "./data.bin"
// #define OUTFILE "./snapshot.bin"

// page size is 32bytes
#define PAGE_SIZE (1 << 5)
// 16 KB in page table
#define INVERT_PAGE_TABLE_SIZE (1 << 14)
// 32 KB in shared memory
#define PHYSICAL_MEM_SIZE (1 << 15)
// 128 KB in global memory
#define STORAGE_SIZE (1 << 17)

//// count the pagefault times
__device__ __managed__ int pagefault_num = 0;

// data input and output
__device__ __managed__ uchar results[4][STORAGE_SIZE];
__device__ __managed__ uchar input[STORAGE_SIZE];

// memory allocation for virtual_memory
// secondary memory
__device__ __managed__ uchar storage[STORAGE_SIZE];
// page table
extern __shared__ u32 pt[];
__device__ __managed__ int priority = 0;

__device__ void user_program(VirtualMemory *vm, uchar *input, uchar *results,
							 int input_size);
__host__ void write_binaryFile(char *fileName, void *buffer, int bufferSize);

__global__ void mykernel(int input_size)
{

	int thread_id = getLocalThreadId();

	__shared__ uchar data[PHYSICAL_MEM_SIZE]; // 32KB-data access in share memory

	// __shared__ int priority;

	if (thread_id == 0)
	{
		priority = 0;
	}
	__syncthreads();

	while (1)
	{
		if (thread_id == priority)
		{

			printf("Thread Id: %d \n", thread_id);
			// memory allocation for virtual_memory
			// take shared memory as physical memory

			VirtualMemory vm;
			vm_init(&vm, data, storage, pt, &pagefault_num, PAGE_SIZE,
					INVERT_PAGE_TABLE_SIZE, PHYSICAL_MEM_SIZE, STORAGE_SIZE,
					PHYSICAL_MEM_SIZE / PAGE_SIZE);
			user_program(&vm, input, results[thread_id], input_size);

			init_LRU(&vm);
			// clear the LRU ?
			printf("input size: %d\n", input_size);
			printf("pagefault number is %d\n", pagefault_num);
			priority++;
			// printf("priority: %d\n", priority);
			break;
		}
		else if (priority > 3)
			break;
		__syncthreads();
	}
}

__host__ void write_binaryFile(char *fileName, void *buffer, int bufferSize)
{
	FILE *fp;
	fp = fopen(fileName, "wb");
	fwrite(buffer, 1, bufferSize, fp);
	fclose(fp);
}

__host__ int load_binaryFile(char *fileName, void *buffer, int bufferSize)
{
	FILE *fp;

	fp = fopen(fileName, "rb");
	if (!fp)
	{
		printf("***Unable to open file %s***\n", fileName);
		exit(1);
	}

	// Get file length
	fseek(fp, 0, SEEK_END);
	int fileLen = ftell(fp);
	fseek(fp, 0, SEEK_SET);

	if (fileLen > bufferSize)
	{
		printf("****invalid testcase!!****\n");
		printf("****software warrning: the file: %s size****\n", fileName);
		printf("****is greater than buffer size****\n");
		exit(1);
	}

	// Read file contents into buffer
	fread(buffer, fileLen, 1, fp);
	fclose(fp);

	return fileLen;
}

int main()
{
	hipError_t cudaStatus;

	int input_size = load_binaryFile(DATAFILE, input, STORAGE_SIZE);

	char output_file1[] = "snapshot_1.bin";
	char output_file2[] = "snapshot_2.bin";
	char output_file3[] = "snapshot_3.bin";
	char output_file4[] = "snapshot_4.bin";
	char *output_file[4] = {output_file1, output_file2, output_file3, output_file4};
	// user program the access pattern for testing paging
	/* Launch kernel function in GPU, with single thread
	and dynamically allocate INVERT_PAGE_TABLE_SIZE bytes of share memory,
	which is used for variables declared as "extern __shared__" */
	mykernel<<<1, 4, INVERT_PAGE_TABLE_SIZE>>>(input_size);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "mykernel launch failed: %s\n",
				hipGetErrorString(cudaStatus));
		return;
	}

	for (int i = 0; i < 4; i++)
	{
		write_binaryFile(output_file[i], results[i], input_size);
	}
	hipDeviceSynchronize();
	hipDeviceReset();
	return 0;
}
